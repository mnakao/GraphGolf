#include "hip/hip_runtime.h"
#include "common.h"
uint64_t *A_dev, *B_dev;
uint64_t *result, *result_dev;
int *adjacency_dev;

__global__ void matrix_op_init_dev(uint64_t* __restrict__ A, uint64_t* __restrict__ B,
                                   const int based_nodes, const int t, const int chunk)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid<UINT64_BITS*chunk && UINT64_BITS*t*chunk+tid<based_nodes) {
    unsigned int offset = (UINT64_BITS*t*chunk+tid)*chunk+tid/UINT64_BITS;
    A[offset] = B[offset] = (0x1ULL<<(tid%UINT64_BITS));
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void clear_buffers_dev(uint64_t* __restrict__ A, uint64_t* __restrict__ B,
				  const int length)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid<length) {
    A[tid] = B[tid] = 0;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ static void matrix_op_dev(const uint64_t* __restrict__ A, uint64_t* __restrict__ B,
				     const int* __restrict__ adjacency, const int nodes,
				     const int degree, const unsigned int elements)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < nodes*elements) {
    int i = tid / elements;
    int k = tid % elements;
    uint64_t tmp = B[tid];
    for(int j=0;j<degree;j++){
      int n = *(adjacency + i * degree + j);  // int n = adjacency[i][j];
      tmp |= A[n*elements+k];
    }
    B[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ static void popcnt_dev(const uint64_t* __restrict__ B, const int nodes, 
				  const unsigned int elements, uint64_t* __restrict__ result)
{
  __shared__ uint64_t cache[THREADS];
  int cacheIndex = threadIdx.x;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  uint64_t num = 0;
  while (tid < elements*nodes) {
    num += POPCNT(B[tid]);
    tid += blockDim.x * gridDim.x;
  }
  cache[cacheIndex] = num;
  __syncthreads();

  int i = blockDim.x/2;
  while (i != 0){
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex+i];
    __syncthreads();
    i /= 2;
  }

  if(cacheIndex == 0)
    result[blockIdx.x] = cache[0];
}

extern "C" bool matrix_op(const int nodes, const int degree, const int based_nodes,
			  const int* __restrict__ adjacency, const int groups,
			  int *diameter, double *ASPL)
{
  unsigned int elements = (based_nodes+UINT64_BITS-1)/UINT64_BITS;
  unsigned int chunk = (elements+(procs-1))/procs;
  int parsize = (elements + chunk - 1)/chunk;

  double sum = 0.0;
  *diameter = 1;
  hipMemcpy(adjacency_dev, adjacency, sizeof(int)*nodes*degree, hipMemcpyHostToDevice);
  
  for(int t=rank;t<parsize;t+=procs){
    unsigned int kk, l;
    for(l=0; l<UINT64_BITS*chunk && UINT64_BITS*t*chunk+l<based_nodes; l++){}
    
    clear_buffers_dev  <<< BLOCKS, THREADS >>> (A_dev, B_dev, nodes*chunk);
    matrix_op_init_dev <<< BLOCKS, THREADS >>> (A_dev, B_dev, based_nodes, t, chunk);
  
    for(kk=0;kk<nodes;kk++){
      matrix_op_dev <<< BLOCKS, THREADS >>> (A_dev, B_dev, adjacency_dev,
					     nodes, degree, chunk);
      popcnt_dev <<< BLOCKS, THREADS >>> (B_dev, nodes, chunk, result_dev);

      hipMemcpy(result, result_dev, sizeof(uint64_t)*BLOCKS, hipMemcpyDeviceToHost);
      uint64_t num = 0;
      for (int i=0;i<BLOCKS;i++)
	num += result[i];

      if(num == (uint64_t)nodes*l) break;

      // swap A <-> B
      uint64_t* tmp = A_dev;
      A_dev = B_dev;
      B_dev = tmp;
    
      sum += ((double)nodes * l - num) * groups;
    }
    *diameter = MAX(*diameter, kk+1);
  }
  MPI_Allreduce(MPI_IN_PLACE, diameter, 1, MPI_INT,    MPI_MAX, MPI_COMM_WORLD);
  MPI_Allreduce(MPI_IN_PLACE, &sum,     1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  sum += (double)nodes * (nodes - 1);
  *ASPL = sum / (((double)nodes-1)*nodes);

  if(*diameter > nodes){
    //    ERROR("This graph is not connected graph.\n");
    return false;
  }
  
  return true;
}

extern "C" bool matrix_op_mem_saving(const int nodes, const int degree, const int based_nodes,
				     const int* __restrict__ adjacency,
				     const int groups, int *diameter, double *ASPL)
{
  unsigned int elements = (based_nodes+UINT64_BITS-1)/UINT64_BITS;
  int parsize = (elements+(CHUNK-1))/CHUNK;

  double sum = 0.0;
  *diameter = 1;
  hipMemcpy(adjacency_dev, adjacency, sizeof(int)*nodes*degree, hipMemcpyHostToDevice);

  for(int t=rank;t<parsize;t+=procs){
    unsigned int kk, l;
    for(l=0; l<UINT64_BITS*CHUNK && UINT64_BITS*t*CHUNK+l<based_nodes; l++){}
    
    clear_buffers_dev  <<< BLOCKS, THREADS >>> (A_dev, B_dev, nodes*CHUNK);
    matrix_op_init_dev <<< BLOCKS, THREADS >>> (A_dev, B_dev, based_nodes, t, CHUNK);

    for(kk=0;kk<nodes;kk++){
      matrix_op_dev <<< BLOCKS, THREADS >>> (A_dev, B_dev, adjacency_dev, nodes, degree, CHUNK);
      popcnt_dev <<< BLOCKS, THREADS >>> (B_dev, nodes, CHUNK, result_dev);
      
      hipMemcpy(result, result_dev, sizeof(uint64_t)*BLOCKS, hipMemcpyDeviceToHost);
      uint64_t num = 0;
      for (int i=0;i<BLOCKS;i++)
	num += result[i];

      if(num == (uint64_t)nodes*l) break;

      // swap A <-> B
      uint64_t* tmp = A_dev;
      A_dev = B_dev;
      B_dev = tmp;

      sum += ((double)nodes * l - num) * groups;
    }
    *diameter = MAX(*diameter, kk+1);
  }
  MPI_Allreduce(MPI_IN_PLACE, diameter, 1, MPI_INT,    MPI_MAX, MPI_COMM_WORLD);
  MPI_Allreduce(MPI_IN_PLACE, &sum,     1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  sum += (double)nodes * (nodes - 1);
  *ASPL = sum / (((double)nodes-1)*nodes);

  if(*diameter > nodes){
    //    ERROR("This graph is not connected graph.\n");
    return false;
  }

  return true;
}

extern "C" void init_matrix_dev(const int nodes, const int degree, const int based_nodes,
				const int algo)
{
  hipInit(0);
  int gpus = -1;
  hipGetDeviceCount(&gpus);
  hipSetDevice(rank%gpus);
  unsigned int elements = (based_nodes+UINT64_BITS-1)/UINT64_BITS;
  size_t s = (algo == MATRIX_OP)? (elements+procs-1)/procs : CHUNK;
  s *= nodes * sizeof(uint64_t);

  hipMalloc((void**)&A_dev, s);
  hipMalloc((void**)&B_dev, s);
  hipHostAlloc((void**)&result, BLOCKS*sizeof(uint64_t), hipHostMallocDefault);
  hipMalloc((void**)&result_dev,      sizeof(uint64_t)*BLOCKS);
  hipMalloc((void**)&adjacency_dev,   sizeof(int)*nodes*degree);
}

extern "C" void finalize_matrix_dev()
{
  hipFree(A_dev);
  hipFree(B_dev);
  hipHostFree(result);
  hipFree(result_dev);
  hipFree(adjacency_dev);
}

extern "C" bool evaluation(const int nodes, const int based_nodes, const int groups, const int lines, 
			   const int degree, int* __restrict__ adjacency, int* __restrict__ diameter, 
			   double* __restrict__ ASPL, const int added_centers, const int algo)
{
  timer_start(TIMER_APSP);
  bool ret;
  if(algo == MATRIX_OP)
    ret = matrix_op(nodes, degree, based_nodes, adjacency, groups, diameter, ASPL);
  else // algo == MATRIX_OP_MEM_SAVING
    ret = matrix_op_mem_saving(nodes, degree, based_nodes, adjacency, groups, diameter, ASPL);
  
  timer_stop(TIMER_APSP);
  return ret;
}
